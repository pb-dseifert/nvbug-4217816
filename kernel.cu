#include "hip/hip_runtime.h"
// TO "FIX" THE BUG, DO ONE OF THE FOLLOWING:
//
// 1. comment out the #include "kernel.cuh" on line 7
// 2. or remove the top-level, right-most const in the MyKernel definition on line 12
// 3. or add the top-level, right-most const in the MyKernel declaration on line 2 in kernel.cuh

#include "kernel.cuh"

#include <stdio.h>

template <typename T>
__global__ void MyKernel(const void* const) { printf("Hello World\n"); }

template __global__ void MyKernel<int>(const void*);
